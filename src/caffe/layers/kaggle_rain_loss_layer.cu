#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void KaggleRainLossForwardGPU(const int nthreads,
          const Dtype* bottom_data, const Dtype* h_func_data) {

  CUDA_KERNEL_LOOP(i, nthreads) {
      int rain  = (int)(bottom_data[i]);
      const Dtype d0(0); 
      const Dtype d1(1); 
      caffe_gpu_set(rain, d0, h_func_data + i*70);
      caffe_gpu_set(70 - rain, d1, h_func_data + i*70 + rain);
  }

}

template <typename Dtype>
void KaggleRainLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  int num = bottom[0]->num();
  int nthreads = num;
  Dtype* h_func_data = h_func_.mutable_gpu_data();

//  KaggleRainLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
//      CAFFE_CUDA_NUM_THREADS>>>(nthreads, bottom[1].gpu_data(), h_func_data);

  for (int i = 0; i < num; i ++) {
      int rain  = bottom[1]->data_at(i, 0, 0, 0);
      caffe_gpu_set(rain, Dtype(0), h_func_data + i*70);
      caffe_gpu_set(70 - rain, Dtype(1), h_func_data + i*70 + rain);
  }

  int count = bottom[0]->count();
  caffe_gpu_sub(
      count, 
      bottom[0]->gpu_data(), 
      h_func_.gpu_data(), 
      diff_.mutable_gpu_data());

  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  Dtype loss = dot / bottom[0]->num() / Dtype(70);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void KaggleRainLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  if (propagate_down[0]) {
      const Dtype alpha = top[0]->cpu_diff()[0] / bottom[0]->num();
      caffe_gpu_axpby(
          bottom[0]->count(),              // count
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[0]->mutable_gpu_diff());  // b
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(KaggleRainLossLayer);

}  // namespace caffe
