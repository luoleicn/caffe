#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void KaggleRainLossForwardGPU(const int nthreads,
          const Dtype* bottom_data, const Dtype* h_func_data) {

  CUDA_KERNEL_LOOP(i, nthreads) {
      int rain  = (int)(bottom_data[i]);
      const Dtype d0(0); 
      const Dtype d1(1); 
      caffe_gpu_set(rain, d0, h_func_data + i*70);
      caffe_gpu_set(70 - rain, d1, h_func_data + i*70 + rain);
  }

}

template <typename Dtype>
void KaggleRainLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  int num = bottom[0]->num();
  //int nthreads = num;
  Dtype* h_func_data = h_func_.mutable_gpu_data();

//  KaggleRainLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
//      CAFFE_CUDA_NUM_THREADS>>>(nthreads, bottom[1].gpu_data(), h_func_data);

  for (int i = 0; i < num; i ++) {
      int rain  = bottom[1]->data_at(i, 0, 0, 0);
      caffe_gpu_set(rain, Dtype(0), h_func_data + i*70);
      caffe_gpu_set(70 - rain, Dtype(1), h_func_data + i*70 + rain);
  }

  Dtype* cdf = cdf_.mutable_cpu_data();
  memcpy(cdf, bottom[0]->cpu_data(), bottom[0]->count()*sizeof(Dtype));

  for (int i = 0; i < num; i ++) {
      Dtype last(0);
      for (int j = 0; j < 70; j ++) {
          cdf[i*70+j] += last;
          last = cdf[i*70+j];
      }
  }

  int count = bottom[0]->count();
  caffe_gpu_sub(
      count, 
      cdf_.gpu_data(), 
      h_func_.gpu_data(), 
      diff_.mutable_gpu_data());

  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  Dtype loss = dot / bottom[0]->num() / Dtype(70);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void KaggleRainLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  if (propagate_down[0]) {
      const Dtype alpha = top[0]->cpu_diff()[0] / bottom[0]->num() / Dtype(35);
      caffe_gpu_axpby(
          bottom[0]->count(),              // count
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[0]->mutable_gpu_diff());  // b

      Dtype* ret_diff = bottom[0]->mutable_cpu_diff();
      Dtype last(0);
      for (int i = 69; i >= 0; i --) {
          ret_diff[i] += last;
          last = ret_diff[i];
      }

  }
}

INSTANTIATE_LAYER_GPU_FUNCS(KaggleRainLossLayer);

}  // namespace caffe
