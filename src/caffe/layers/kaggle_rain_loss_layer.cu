#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void KaggleRainLossForwardGPU(const int nthreads,
          const Dtype* bottom_data, const Dtype* h_func_data) {

  CUDA_KERNEL_LOOP(i, nthreads) {
      int rain  = (int)(bottom_data[i]);
      const Dtype d0(0); 
      const Dtype d1(1); 
      caffe_gpu_set(rain, d0, h_func_data + i*70);
      caffe_gpu_set(70 - rain, d1, h_func_data + i*70 + rain);
  }

}

template <typename Dtype>
void KaggleRainLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  int num = bottom[0]->num();
  Dtype* h_func_data = h_func_.mutable_gpu_data();

  for (int i = 0; i < num; i ++) {
      int rain  = bottom[1]->data_at(i, 0, 0, 0);
      //LOG(INFO) << "rain i=" << i << " rain=" << rain;
      caffe_gpu_set(rain, Dtype(0), h_func_data + i*70);
      caffe_gpu_set(70 - rain, Dtype(1), h_func_data + i*70 + rain);
  }

  Dtype* cdf = cdf_.mutable_cpu_data();
  //memset(cdf, Dtype(1), cdf->count()*sizeof(Dtype));
  for (int i = 0; i < num; i ++) {
      Dtype last(0);
      for (int j = 0; j < 70; j ++) {
	      if (j <= 10) {
		      cdf[i*70+j] = bottom[0]->data_at(i, j, 0, 0) + last;
		      last = cdf[i*70+j];
	      }
	      else {
		      cdf[i*70+j] = Dtype(1);
	      }
	      //LOG(INFO) << "i=" << i << " j=" << j << " cdf=" << cdf[i*70+j];
      }
  }


  int count = cdf_.count();
  caffe_gpu_sub(
      count, 
      cdf_.gpu_data(), 
      h_func_.gpu_data(), 
      diff_.mutable_gpu_data());

//  for (int i = 0; i < num; i ++) {
//      for (int j = 0; j < 70; j ++) {
//	      if (j <= 10) {
//		      LOG(INFO) << "i=" << i << " j=" << j << " cdf=" << cdf[i*70+j]
//			      << " h_func_=" << h_func_.data_at(i, j, 0, 0)
//			      << " diff=" << diff_.data_at(i, j, 0, 0);
//	      }
//      }
//  }

  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  Dtype loss = dot / bottom[0]->num() / Dtype(70);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void KaggleRainLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  if (propagate_down[0]) {
      const Dtype alpha = top[0]->cpu_diff()[0] / bottom[0]->num() / Dtype(35);
      //caffe_gpu_axpby(
      //    bottom[0]->count(),              // count
      //    alpha,                              // alpha
      //    diff_.gpu_data(),                   // a
      //    Dtype(0),                           // beta
      //    bottom[0]->mutable_gpu_diff());  // b

      Dtype* ret_diff = bottom[0]->mutable_cpu_diff();
      int num = bottom[0]->num();
      for (int i = 0; i < num; i ++) {
	      Dtype last(0);
	      for (int j = 69; j >= 0; j --) {
		      if (j > 10) {
			      continue;
		      }
		      else {
			      ret_diff[i*10+j] = last + diff_.data_at(i, j, 0, 0);
			      ret_diff[i*10+j] *= alpha;

			      last += diff_.data_at(i, j, 0, 0);
		      }
	      }
      }

  }
}

INSTANTIATE_LAYER_GPU_FUNCS(KaggleRainLossLayer);

}  // namespace caffe
