#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void KaggleRainLossForwardGPU(const int nthreads,
          const Dtype* bottom_data, const Dtype* h_func_data) {

  CUDA_KERNEL_LOOP(i, nthreads) {
      int rain  = (int)(bottom_data[i]);
      const Dtype d0(0); 
      const Dtype d1(1); 
      caffe_gpu_set(rain, d0, h_func_data + i*71);
      caffe_gpu_set(71 - rain, d1, h_func_data + i*71 + rain);
  }

}

template <typename Dtype>
void KaggleRainLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {

  const int LEARNED_CLASS = 11;

  int num = bottom[0]->num();
  Dtype* bottom_data = bottom[0]->mutable_cpu_data();
  for (int i = 0; i < num; i ++) {
      Dtype sum(0);
      for (int j = 0; j < LEARNED_CLASS; j ++) {
	      sum += bottom_data[i*LEARNED_CLASS + j];
      }
      for (int j = 0; j < LEARNED_CLASS; j ++) {
	      if (sum == 0) {
		      //LOG(INFO) << "i=" << i << " j=" << j << " prob " << bottom_data[i*LEARNED_CLASS + j]; 
		      bottom_data[i*LEARNED_CLASS + j] = 1.0 / LEARNED_CLASS;
	      }
	      else {
		      bottom_data[i*LEARNED_CLASS + j] /= sum;
	      }
      }
  }
  Dtype* h_func_data = h_func_.mutable_gpu_data();

  for (int i = 0; i < num; i ++) {
      int rain  = bottom[1]->data_at(i, 0, 0, 0);
      //LOG(INFO) << "rain i=" << i << " rain=" << rain;
      caffe_gpu_set(rain, Dtype(0), h_func_data + i*71);
      caffe_gpu_set(71 - rain, Dtype(1), h_func_data + i*71 + rain);
  }

  Dtype* cdf = cdf_.mutable_cpu_data();
  //memset(cdf, Dtype(1), cdf->count()*sizeof(Dtype));
  for (int i = 0; i < num; i ++) {
      Dtype last(0);
      for (int j = 0; j < 71; j ++) {
	      if (j == 70) {
		      cdf[i*71+j] = Dtype(1);
	      }
	      else if (j < LEARNED_CLASS) {
		      cdf[i*71+j] = bottom[0]->data_at(i, j, 0, 0) + last;
		      last = cdf[i*71+j];
	      }
	      else {
		      cdf[i*71+j] = Dtype(1);
	      }
	      //LOG(INFO) << "i=" << i << " j=" << j << " cdf=" << cdf[i*70+j];
      }
  }


  int count = cdf_.count();
  caffe_gpu_sub(
      count, 
      cdf_.gpu_data(), 
      h_func_.gpu_data(), 
      diff_.mutable_gpu_data());


  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  Dtype loss = dot / bottom[0]->num() / Dtype(70);
  top[0]->mutable_cpu_data()[0] = loss;
  
  //if (loss < 0.005) {
  //        for (int i = 0; i < num; i ++) {
  //      	  for (int j = 0; j < 70; j ++) {
  //      		  if (j <= 10) {
  //      			  LOG(INFO) << "i=" << i << " j=" << j << " cdf=" << cdf[i*71+j]
  //      				  << " h_func_=" << h_func_.data_at(i, j, 0, 0)
  //      				  << " diff=" << diff_.data_at(i, j, 0, 0);
  //      		  }
  //      	  }
  //        }
  //        LOG(INFO) << " loss " << loss << " dot" << dot;
  //        exit(-1);
  //}
}

template <typename Dtype>
void KaggleRainLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

	const int LEARNED_CLASS = 11;
  if (propagate_down[0]) {
      const Dtype alpha = top[0]->cpu_diff()[0] / bottom[0]->num() / Dtype(35);
      //caffe_gpu_axpby(
      //    bottom[0]->count(),              // count
      //    alpha,                              // alpha
      //    diff_.gpu_data(),                   // a
      //    Dtype(0),                           // beta
      //    bottom[0]->mutable_gpu_diff());  // b

      Dtype* ret_diff = bottom[0]->mutable_cpu_diff();
      int num = bottom[0]->num();
      memset(ret_diff, 0, sizeof(Dtype) * num * LEARNED_CLASS);
      for (int i = 0; i < num; i ++) {
	      Dtype last(0);
	      for (int j = 70; j >= 0; j --) {
		      if (j >= LEARNED_CLASS) {
			      continue;
		      }
		      else {
			      ret_diff[i*LEARNED_CLASS+j] = last + diff_.data_at(i, j, 0, 0);
			      ret_diff[i*LEARNED_CLASS+j] *= alpha;

			      last += diff_.data_at(i, j, 0, 0);
		      }
	      }
      }

  }
}

INSTANTIATE_LAYER_GPU_FUNCS(KaggleRainLossLayer);

}  // namespace caffe
